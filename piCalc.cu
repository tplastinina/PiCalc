#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
// thrust library
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/generate.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "ctime"


__global__ void calcPi(float *x, float *y, int *blocks_counts, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = gridDim.x * blockDim.x;
	__shared__ int counts[512];
	int count = 0;
	for (int i = idx; i < n; i += offset) {
		if (x[i] * x[i] + y[i] * y[i] < 1.0f) {
			count++;
		}
	}
	counts[threadIdx.x] = count;

	__syncthreads();

	if (threadIdx.x == 0) {
		int total = 0;
		for (int j = 0; j < 512; j++) {
			total += counts[j];
		}
		blocks_counts[blockIdx.x] = total;
	}
}


void gpu_fillRand(float *a, float *b, unsigned int size) {
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
	hiprandGenerateUniform(prng, a, size);
	hiprandGenerateUniform(prng, b, size);
}

float calcPICPU(float *x, float *y, unsigned size){
	int count = 0;
	for (int i = 0; i < size; i++) {
		if (x[i] * x[i] + y[i] * y[i] < 1.0f) {
			count++;
		}
	}
	return float(count) * 4.0 / size;
}

int main()
{
	long n = 1024* 1024 * 16 * 2;

	float *hostX, *hostY, *devX, *devY;


	hostX = (float *)calloc(n, sizeof(float));
	hostY = (float *)calloc(n, sizeof(float));

	hipMalloc((void **)&devX, n * sizeof(float));
	hipMalloc((void **)&devY, n * sizeof(float));


	float* ptrX = thrust::raw_pointer_cast(&devX[0]);
	float* ptrY = thrust::raw_pointer_cast(&devY[0]);


	gpu_fillRand(ptrX, ptrY, n);

	hipMemcpy(hostX, ptrX, n * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hostY, ptrY, n * sizeof(float), hipMemcpyDeviceToHost);

	clock_t startCPU;
	startCPU = clock();
	float cpu_result = calcPICPU(hostX, hostY, n);
	printf("\nCPU's time for pi: %f", (clock() - startCPU) / (double)CLOCKS_PER_SEC);
	printf("\nCPU's result: %f", cpu_result);


	int *dev_blocks_counts = 0, *blocks_counts = 0;
	float gpuTime = 0.0f;
	hipEvent_t start, stop;

	int blockDim = 512;
	int gridDim = n / (128 * blockDim);

	blocks_counts = (int *)calloc(gridDim, sizeof(int));

	hipMalloc((void **)&dev_blocks_counts, 512 * sizeof(int));
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMemset(dev_blocks_counts, 0, sizeof(int));

	calcPi<<<gridDim, blockDim>>>(devX, devY, dev_blocks_counts, n);

	hipMemcpy(blocks_counts, dev_blocks_counts, gridDim * sizeof(int), hipMemcpyDeviceToHost);
	int count = 0;
	for (int i = 0; i < gridDim; i++) {
		count += blocks_counts[i];
	};

	float gpu_result = float(count) * 4 / float(n);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("\nGPU's time spent executing %s: %f seconds", "kernel", gpuTime / 1000);
	printf("\nGPU's result: %f\n", gpu_result);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(ptrX);
	hipFree(ptrY);
	hipFree(dev_blocks_counts);
	return 0;
}
